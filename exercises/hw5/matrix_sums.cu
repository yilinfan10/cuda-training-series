
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>
#include <ctime>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


const size_t DSIZE = 16384;      // matrix side dimension
const int block_size = 256;  // CUDA maximum is 1024

 __device__ void reduce_a(const float *gdata, float *out){
     __shared__ float sdata[block_size];
     int tid = threadIdx.x;
     sdata[tid] = 0.0f;
     size_t idx = threadIdx.x+blockDim.x*blockIdx.x;

     while (idx < DSIZE) {  // grid stride loop to load data
        sdata[tid] += gdata[idx];
        idx += gridDim.x*blockDim.x;  
        }

     for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        __syncthreads();
        if (tid < s)  // parallel sweep reduction
            sdata[tid] += sdata[tid + s];
        }
     if (tid == 0) atomicAdd(out, sdata[0]);
  }
// matrix row-sum kernel
  __global__ void row_sums(const float* A, float* sums, size_t ds) {
    int row_step = (blockDim.x * gridDim.x) / warpSize;
    int col_step = warpSize;
    unsigned mask = 0xFFFFFFFFU;

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int col = idx % col_step;
    int lane = col;
    for (int row = idx / col_step; row < ds; row += row_step) {
      int row_offset = row * ds;
      float val = 0.0f;
      col = lane;
      while (col < ds) {
        val += A[row_offset + col];
        col += col_step;
      }
      for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(mask, val, offset);
      }
      if (lane == 0) {
        sums[row] = val;
      }
    }
  }
// matrix column-sum kernel
__global__ void column_sums(const float *A, float *sums, size_t ds){

  int idx = threadIdx.x+blockDim.x*blockIdx.x; // create typical 1D thread index from built-in variables
  if (idx < ds){
    float sum = 0.0f;
    for (size_t i = 0; i < ds; i++)
      sum += A[idx+ds*i];         // write a for loop that will cause the thread to iterate down a column, keeeping a running sum, and write the result to sums
    sums[idx] = sum;
}}
bool validate(float *data, size_t sz){
  for (size_t i = 0; i < sz; i++)
    if (data[i] != (float)sz) {printf("results mismatch at %lu, was: %f, should be: %f\n", i, data[i], (float)sz); return false;}
    return true;
}
int main(){

  float *h_A, *h_sums, *d_A, *d_sums;
  h_A = new float[DSIZE*DSIZE];  // allocate space for data in host memory
  h_sums = new float[DSIZE]();
  for (int i = 0; i < DSIZE*DSIZE; i++)  // initialize matrix in host memory
    h_A[i] = 1.0f;
  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));  // allocate device space for A
  hipMalloc(&d_sums, DSIZE*sizeof(float));  // allocate device space for vector d_sums
  cudaCheckErrors("hipMalloc failure"); // error checking
  // copy matrix A to device:
  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");
  //cuda processing sequence step 1 is complete
  clock_t t = clock();
  row_sums<<<(DSIZE+block_size-1)/block_size, block_size>>>(d_A, d_sums, DSIZE);
  cudaCheckErrors("kernel launch failure");
  hipStreamSynchronize(nullptr);
  printf("%f\n", static_cast<float>(clock() - t) / CLOCKS_PER_SEC);
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sums, d_sums, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  if (!validate(h_sums, DSIZE)) return -1; 
  printf("row sums correct!\n");
  hipMemset(d_sums, 0, DSIZE*sizeof(float));
  t = clock();
  column_sums<<<(DSIZE+block_size-1)/block_size, block_size>>>(d_A, d_sums, DSIZE);
  cudaCheckErrors("kernel launch failure");
  hipStreamSynchronize(nullptr);
  printf("%f\n", static_cast<float>(clock() - t) / CLOCKS_PER_SEC);
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sums, d_sums, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  if (!validate(h_sums, DSIZE)) return -1; 
  printf("column sums correct!\n");
  return 0;
}
  
